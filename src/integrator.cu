#include "hip/hip_runtime.h"
#include "integrator.hpp"
#include "randomvariable.hpp"
#include "kernel.hpp"
#include "config.hpp"

#include <hiprand.h>

#include <iostream>
#include <fstream>
#include <math.h>


Integrator::~Integrator()
{
    hiprandDestroyGenerator( *(this->gen) );
}


Integrator::Integrator()
{
    size_t size = sizeof(float) * INTEGRATION_SAMPLES;

    this->gen = new hiprandGenerator_t();
    hiprandCreateGenerator( this->gen, HIPRAND_RNG_PSEUDO_MTGP32 );
    hiprandSetPseudoRandomGeneratorSeed( *(this->gen), 1234ULL );
}


float
Integrator::distance (TimeSeries &ts1, TimeSeries &ts2, int n)
{
    size_t seq_size  = sizeof(float) * n * PARAM_SIZE;
    size_t dust_size = sizeof(float) * n;

    // copy ts1, ts2 to memory
    float *seq, *dust, *seq_GPU, *dust_GPU;
    seq  = (float*)malloc(seq_size);
    dust = (float*)malloc(dust_size);
    hipMalloc((void**)&seq_GPU, seq_size);
    hipMalloc((void**)&dust_GPU, dust_size);

    for (int i = 0; i < n; i++) {
        RandomVariable x = ts1.at(i);
        RandomVariable y = ts2.at(i);

        int j = PARAM_SIZE * i;
        seq[j]   = (float)x.distribution;
        seq[j+1] = x.observation;
        seq[j+2] = x.stddev;
        seq[j+3] = (float)y.distribution;
        seq[j+4] = y.observation;
        seq[j+5] = y.stddev;
    }

    hipMemcpy( seq_GPU,
                seq,
                seq_size,
                hipMemcpyHostToDevice );

    // generate uniform random number on samples_GPU
    float *samples_GPU;
    hipMalloc( (void**)&samples_GPU, sizeof(float) * INTEGRATION_SAMPLES * n * 3);
    hiprandGenerateUniform( *(this->gen), samples_GPU, INTEGRATION_SAMPLES * n * 3 );


    // call kernel
    g_distance_kernel<<< n, TPB >>>(seq_GPU, samples_GPU, dust_GPU);

    hipMemcpy( dust,
                dust_GPU,
                dust_size,
                hipMemcpyDeviceToHost );


    float dist = 0;
    for (int i=0; i < n; i++) {
        dist += dust[i];
    }


    hipFree( seq_GPU );
    hipFree( dust_GPU );
    hipFree( samples_GPU );

    free(seq);
    free(dust);

    return sqrt(dist);
}


void
Integrator::match_naive (TimeSeries &ts, TimeSeriesCollection &db)
{
    // Determine the length of time series.
    unsigned int lim = ts.length();

    for (int i=0; i < db.sequences.size(); i++) {
        lim = min(lim, db.sequences[i].length());
    }

    float distance_min = this->distance(ts, db.sequences[0], lim);
    float i_min = 0;
    for (int i=1; i < db.sequences.size(); i++) {
        float d = this->distance(ts, db.sequences[i], lim);
        if (d < distance_min) {
            distance_min = d;
            i_min = i;
        }
    }

    std::cout << "matched : " << lim << std::endl;
    std::cout << "\t index: " << i_min << ", distance : " << distance_min << std::endl;
}


void
Integrator::match (TimeSeries &ts, TimeSeriesCollection &db)
{
    // Determine the length of time series.
    size_t db_num = db.sequences.size();
    unsigned int lim = ts.length();
    for (int i=0; i < db_num; i++) {
        lim = min(lim, db.sequences[i].length());
    }

    // db needs (3 * float * lim * db_num) bytes = 3*4*150*200
    // can settle on global memory.
    size_t db_size = sizeof(float) * 3 * db_num * lim;
    float *db_CPU, *db_GPU;
    db_CPU = (float*)malloc(db_size);
    hipMalloc((void**)&db_GPU, db_size);

    size_t dust_size = sizeof(float) * db_num * lim;
    float *dust_CPU, *dust_GPU;
    dust_CPU = (float*)malloc(dust_size);
    hipMalloc((void**)&dust_GPU, dust_size);

    int idx = 0;
    for (int i = 0; i < lim; i++) {
        for (int j = 0; j < db_num; j++) {
            RandomVariable x = db.sequences[j].at(i);
            db_CPU[idx++] = (float)x.distribution;
            db_CPU[idx++] = x.observation;
            db_CPU[idx++] = x.stddev;
            dust_CPU[i*db_num + j] = 0.0f;
        }
    }

    size_t ts_size = sizeof(float) * lim * 3;
    float *ts_CPU, *ts_GPU;
    ts_CPU = (float*)malloc(ts_size);
    hipMalloc((void**)&ts_GPU, ts_size);
    for (int i = 0; i < lim; i++) {
        RandomVariable x = ts.at(i);
        ts_CPU[i]     = (float)x.distribution;
        ts_CPU[i + 1] = x.observation;
        ts_CPU[i + 2] = x.stddev;
    }

    hipMemcpy( db_GPU,
                db_CPU,
                db_size,
                hipMemcpyHostToDevice );

    hipMemcpy( ts_GPU,
                ts_CPU,
                ts_size,
                hipMemcpyHostToDevice );

    hipMemcpy( dust_GPU,
                dust_CPU,
                dust_size,
                hipMemcpyHostToDevice );

    size_t o_size = sizeof(float) * TPB * db_num * lim;
    float *o1, *o2, *o3;
    hipMalloc((void**)&o1, o_size);
    hipMalloc((void**)&o2, o_size);
    hipMalloc((void**)&o3, o_size);

    // generate uniform random number on samples_GPU
    float *samples_GPU;
    hipMalloc( (void**)&samples_GPU, sizeof(float) * INTEGRATION_SAMPLES * lim * 3);
    hiprandGenerateUniform( *(this->gen), samples_GPU, INTEGRATION_SAMPLES * lim * 3 );

    // DO THE STUFF
    g_match<<< lim, TPB >>>(ts_GPU,
                            db_GPU,
                            dust_GPU,
                            lim,
                            db_num,
                            o1,
                            o2,
                            o3,
                            samples_GPU);

    hipMemcpy( dust_CPU,
                dust_GPU,
                dust_size,
                hipMemcpyDeviceToHost );


    // float dust_min;
    // int i_min;
    float dust_min =0;
    int i_min = 0;

//    for (int i = 0; i < db_num * lim; i++) {
    for (int i = 0; i < 10; i++) {
        std::cout << dust_CPU[i] << std::endl;
    }

//     for (int i = 0; i < db_num; i++) {
//         float dist = 0;
//         for (int j = 0; j < lim; j++) {
//             dist += dust_CPU[db_num * j + i];
//             float d = dust_CPU[db_num * j + i];
//             std::cout << d << std::endl;
//         }

//         float d = sqrt(dist);
// //        std::cout << d << std::endl;
//         if (d < dust_min || i == 0) {
//             dust_min = d;
//             i_min = i;
//         }
//     }

    // std::cout << "db_num : " << db_num << std::endl;
    // std::cout << "lim : " << lim << std::endl;
    std::cout << "matched : " << lim << std::endl;
    std::cout << "\t index: " << i_min << ", distance: " << dust_min << std::endl;

    free(db_CPU);
    hipFree(db_GPU);
    free(dust_CPU);
    hipFree(dust_GPU);
    free(ts_CPU);
    hipFree(ts_GPU);
    hipFree(o1);
    hipFree(o2);
    hipFree(o3);
    hipFree(samples_GPU);
}
