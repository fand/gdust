#include "hip/hip_runtime.h"
#include "integrator.hpp"
#include "randomvariable.hpp"
#include "kernel.hpp"
#include "config.hpp"

#include <cutil.h>
#include <hiprand.h>

#include <iostream>
#include <fstream>
#include <math.h>


Integrator::~Integrator()
{
    hiprandDestroyGenerator( *(this->gen) );
}


Integrator::Integrator()
{
    size_t size = sizeof(float) * INTEGRATION_SAMPLES;
    
    this->gen = new hiprandGenerator_t();
    hiprandCreateGenerator( this->gen, HIPRAND_RNG_PSEUDO_MTGP32 );
    hiprandSetPseudoRandomGeneratorSeed( *(this->gen), 1234ULL );
}


float
Integrator::distance (TimeSeries &ts1, TimeSeries &ts2, int n)
{
    size_t seq_size  = sizeof(float) * n * PARAM_SIZE;
    size_t dust_size = sizeof(float) * n;
    
    // copy ts1, ts2 to memory
    float *seq, *dust, *seq_GPU, *dust_GPU;
    seq  = (float*)malloc(seq_size);
    dust = (float*)malloc(dust_size);
    CUDA_SAFE_CALL(hipMalloc((void**)&seq_GPU, seq_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&dust_GPU, dust_size));

    for (int i = 0; i < n; i++) {
        RandomVariable x = ts1.at(i);
        RandomVariable y = ts2.at(i);
        
        int j = PARAM_SIZE * i;
        seq[j]   = (float)x.distribution;
        seq[j+1] = x.observation;
        seq[j+2] = x.stddev;
        seq[j+3] = (float)y.distribution;
        seq[j+4] = y.observation;
        seq[j+5] = y.stddev;
    }
    
    CUDA_SAFE_CALL( hipMemcpy( seq_GPU,
                                seq,
                                seq_size,
                                hipMemcpyHostToDevice ) );
    
    // generate uniform random number on samples_GPU
    float *samples_GPU;
    CUDA_SAFE_CALL( hipMalloc( (void**)&samples_GPU, sizeof(float) * INTEGRATION_SAMPLES * n * 3) );
    hiprandGenerateUniform( *(this->gen), samples_GPU, INTEGRATION_SAMPLES * n * 3 );

    
    // call kernel
    g_distance_kernel<<< n, TPB >>>(seq_GPU, samples_GPU, dust_GPU);

    CUDA_SAFE_CALL( hipMemcpy( dust,
                                dust_GPU,
                                dust_size,
                                hipMemcpyDeviceToHost ) );


    float dist = 0;
    for (int i=0; i < n; i++) {
        dist += dust[i];
    }


    CUDA_SAFE_CALL( hipFree( seq_GPU ) );
    CUDA_SAFE_CALL( hipFree( dust_GPU ) );
    CUDA_SAFE_CALL( hipFree( samples_GPU ) );
    
    free(seq);
    free(dust);
    
    return sqrt(dist);
}
