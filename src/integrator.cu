#include "hip/hip_runtime.h"
#include "integrator.hpp"
#include "randomvariable.hpp"
#include "kernel.hpp"

#include <math.h>
#include <fstream>

#include <cutil.h>
#include <hiprand.h>

#include <iostream>


#define VERYSMALL 1E-20
#define SQRT3 1.73205081

#define RANGE_VALUE SQRT3*10
#define INTEG_RANGE_MAX 16
#define INTEG_RANGE_MIN -16

#define PARAM_SIZE 6
#define INTEGRATION_SAMPLES 49152

#define TPB 512
#define BPG 96


Integrator::~Integrator()
{
    hiprandDestroyGenerator( *(this->gen) );
}


Integrator::Integrator()
{
    size_t size = sizeof(float) * INTEGRATION_SAMPLES;
    
    this->gen = new hiprandGenerator_t();
    hiprandCreateGenerator( this->gen, HIPRAND_RNG_PSEUDO_MTGP32 );
    hiprandSetPseudoRandomGeneratorSeed( *(this->gen), 1234ULL );
}


float
Integrator::distance (TimeSeries &ts1, TimeSeries &ts2, int n)
{
    size_t seq_size  = sizeof(float) * n * PARAM_SIZE;
    size_t dust_size = sizeof(float) * n;
    
    // copy ts1, ts2 to memory
    float *seq, *dust, *seq_GPU, *dust_GPU;
    seq  = (float*)malloc(seq_size);
    dust = (float*)malloc(dust_size);
    CUDA_SAFE_CALL(hipMalloc((void**)&seq_GPU, seq_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&dust_GPU, dust_size));

    for (int i = 0; i < n; i++) {
        RandomVariable x = ts1.at(i);
        RandomVariable y = ts2.at(i);
        
        int j = PARAM_SIZE * i;
        seq[j]   = (float)x.distribution;
        seq[j+1] = x.observation;
        seq[j+2] = x.stddev;
        seq[j+3] = (float)y.distribution;
        seq[j+4] = y.observation;
        seq[j+5] = y.stddev;
    }
    
    CUDA_SAFE_CALL( hipMemcpy( seq_GPU,
                                seq,
                                seq_size,
                                hipMemcpyHostToDevice ) );
    
    // generate uniform random number on samples_GPU
    float *samples_GPU;
    CUDA_SAFE_CALL( hipMalloc( (void**)&samples_GPU, sizeof(float) * INTEGRATION_SAMPLES * n * 3) );
    hiprandGenerateUniform( *(this->gen), samples_GPU, INTEGRATION_SAMPLES * n * 3 );

    
    // call kernel
    g_distance_kernel<<< n, TPB >>>(seq_GPU, samples_GPU, dust_GPU);

    CUDA_SAFE_CALL( hipMemcpy( dust,
                                dust_GPU,
                                dust_size,
                                hipMemcpyDeviceToHost ) );


    float dist = 0;
    for (int i=0; i < n; i++) {
        dist += dust[i];
    }


    CUDA_SAFE_CALL( hipFree( seq_GPU ) );
    CUDA_SAFE_CALL( hipFree( dust_GPU ) );
    CUDA_SAFE_CALL( hipFree( samples_GPU ) );
    
    free(seq);
    free(dust);
    
    return sqrt(dist);
}
