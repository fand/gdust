#include "hip/hip_runtime.h"
#include "integrator.hpp"
#include "randomvariable.hpp"
#include "kernel.hpp"
#include "config.hpp"
#include "cutil.hpp"

#include <hiprand.h>

#include <iostream>
#include <fstream>
#include <math.h>


Integrator::~Integrator()
{
    hiprandDestroyGenerator( *(this->gen) );
}


Integrator::Integrator()
{
    size_t size = sizeof(float) * INTEGRATION_SAMPLES;

    this->gen = new hiprandGenerator_t();
    hiprandCreateGenerator( this->gen, HIPRAND_RNG_PSEUDO_MTGP32 );
    hiprandSetPseudoRandomGeneratorSeed( *(this->gen), 1234ULL );
}


float
Integrator::distance (TimeSeries &ts1, TimeSeries &ts2, int n)
{
    size_t seq_size  = sizeof(float) * n * PARAM_SIZE;
    size_t dust_size = sizeof(float) * n;

    // copy ts1, ts2 to memory
    float *seq, *dust, *seq_GPU, *dust_GPU;
    seq  = (float*)malloc(seq_size);
    dust = (float*)malloc(dust_size);
    checkCudaErrors(hipMalloc((void**)&seq_GPU, seq_size));
    checkCudaErrors(hipMalloc((void**)&dust_GPU, dust_size));

    for (int i = 0; i < n; i++) {
        RandomVariable x = ts1.at(i);
        RandomVariable y = ts2.at(i);

        int j = PARAM_SIZE * i;
        seq[j]   = (float)x.distribution;
        seq[j+1] = x.observation;
        seq[j+2] = x.stddev;
        seq[j+3] = (float)y.distribution;
        seq[j+4] = y.observation;
        seq[j+5] = y.stddev;
    }

    checkCudaErrors(hipMemcpy( seq_GPU,
                                seq,
                                seq_size,
                                hipMemcpyHostToDevice ));

    // generate uniform random number on samples_GPU
    float *samples_GPU;
    checkCudaErrors(hipMalloc( (void**)&samples_GPU, sizeof(float) * INTEGRATION_SAMPLES * n * 3));
    hiprandGenerateUniform( *(this->gen), samples_GPU, INTEGRATION_SAMPLES * n * 3 );


    // call kernel
    g_distance_kernel<<< n, TPB >>>(seq_GPU, samples_GPU, dust_GPU);

    checkCudaErrors(hipMemcpy( dust,
                                dust_GPU,
                                dust_size,
                                hipMemcpyDeviceToHost ));


    float dist = 0;
    for (int i=0; i < n; i++) {
        dist += dust[i];
    }


    checkCudaErrors(hipFree( seq_GPU ));
    checkCudaErrors(hipFree( dust_GPU ));
    checkCudaErrors(hipFree( samples_GPU ));

    free(seq);
    free(dust);

    return sqrt(dist);
}


void
Integrator::match_naive (TimeSeries &ts, TimeSeriesCollection &db)
{
    // Determine the length of time series.
    unsigned int lim = ts.length();

    for (int i = 0; i < db.sequences.size(); i++) {
        lim = min(lim, db.sequences[i].length());
    }

    float DUST_min;
    float i_min;

    for (int i = 0; i < db.sequences.size(); i++) {
        float DUST = this->distance(ts, db.sequences[i], lim);

        // std::cout << "CPU " << i << ": " << DUST << std::endl;

        if (DUST < DUST_min || i == 0) {
            DUST_min = DUST;
            i_min = i;
        }
    }

    std::cout << "matched : " << lim << std::endl;
    std::cout << "\t index: " << i_min << ", distance : " << DUST_min << std::endl;
}


void
Integrator::match (TimeSeries &ts, TimeSeriesCollection &db)
{
    // Determine the length of time series.
    size_t db_num = db.sequences.size();
    unsigned int lim = ts.length();
    for (int i=0; i < db_num; i++) {
        lim = min(lim, db.sequences[i].length());
    }

    // db needs (3 * float * lim * db_num) bytes =~ 3*4*150*200
    // can settle on global memory.
    size_t db_size = sizeof(float) * 3 * db_num * lim;
    float *db_CPU, *db_GPU;
    db_CPU = (float*)malloc(db_size);
    checkCudaErrors(hipMalloc((void**)&db_GPU, db_size));

    size_t dust_size = sizeof(float) * db_num * lim;
    float *dust_CPU, *dust_GPU;
    dust_CPU = (float*)malloc(dust_size);
    checkCudaErrors(hipMalloc((void**)&dust_GPU, dust_size));

    int idx = 0;
    for (int i = 0; i < lim; i++) {
        for (int j = 0; j < db_num; j++) {
            RandomVariable x = db.sequences[j].at(i);
            db_CPU[idx++] = (float)x.distribution;
            db_CPU[idx++] = x.observation;
            db_CPU[idx++] = x.stddev;
        }
    }

    size_t ts_size = sizeof(float) * lim * 3;
    float *ts_CPU, *ts_GPU;
    ts_CPU = (float*)malloc(ts_size);
    checkCudaErrors(hipMalloc((void**)&ts_GPU, ts_size));
    idx = 0;
    for (int i = 0; i < lim; i++) {
        RandomVariable x = ts.at(i);
        ts_CPU[idx++] = (float)x.distribution;
        ts_CPU[idx++] = x.observation;
        ts_CPU[idx++] = x.stddev;
    }

    checkCudaErrors(hipMemcpy( db_GPU,
                                db_CPU,
                                db_size,
                                hipMemcpyHostToDevice ));

    checkCudaErrors(hipMemcpy( ts_GPU,
                                ts_CPU,
                                ts_size,
                                hipMemcpyHostToDevice ));

    // generate uniform random number on samples_GPU
    float *samples_GPU;
    checkCudaErrors(hipMalloc( (void**)&samples_GPU, sizeof(float) * INTEGRATION_SAMPLES * lim * db_num * 3));
    hiprandGenerateUniform( *(this->gen), samples_GPU, INTEGRATION_SAMPLES * lim * db_num * 3);


    // DO THE STUFF
    g_match<<< lim, TPB >>>( ts_GPU,
                             db_GPU,
                             dust_GPU,
                             lim,
                             db_num,
                             samples_GPU );

    checkCudaErrors(hipMemcpy( dust_CPU,
                                dust_GPU,
                                dust_size,
                                hipMemcpyDeviceToHost ));

    float DUST_min;
    int i_min = 0;

    for (int i = 0; i < db_num; i++) {
        float dist = 0;
        for (int j = 0; j < lim; j++) {
            dist += dust_CPU[db_num * j + i];
        }

        float DUST = sqrt(dist);
        // std::cout << "GPU " << i << ": " << DUST << std::endl;
        if (DUST < DUST_min || i == 0) {
            DUST_min = DUST;
            i_min = i;
        }
    }

    // std::cout << "db_num : " << db_num << std::endl;
    // std::cout << "lim : " << lim << std::endl;
    std::cout << "matched : " << lim << std::endl;
    std::cout << "\t index: " << i_min << ", distance: " << DUST_min << std::endl;

    free(db_CPU);
    free(dust_CPU);
    free(ts_CPU);
    checkCudaErrors(hipFree(db_GPU));
    checkCudaErrors(hipFree(dust_GPU));
    checkCudaErrors(hipFree(ts_GPU));
    checkCudaErrors(hipFree(samples_GPU));
}
