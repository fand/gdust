#include "hip/hip_runtime.h"
// Class for DUST with GPU.
// Actual computation is done by Integrator,
// because it's difficult to separate from GPU management.

#include "GDUST.hpp"
#include <algorithm>
#include "Integrator.hpp"
#include "common.hpp"


inline float
clean_probability(float p) {
  if (p <= 0) p = 0;
  return p;
}

GDUST::GDUST(const TimeSeriesCollection &collection, const Integrator::Method method) {
  this->collection = &collection;
  integrator = Integrator::create(method);
}

GDUST::~GDUST() {}

double
GDUST::distance(const TimeSeries &ts1, const TimeSeries &ts2, const int n) {
  int ts_length = min(ts1.length(), ts2.length());
  ts_length = (n == -1) ? ts_length : min(ts_length, n);
  return static_cast<double>(integrator->distance(ts1, ts2, ts_length));
}

int
GDUST::match_naive(const TimeSeries &ts) {
  return this->integrator->match_naive(ts, *this->collection);
}

int
GDUST::match(const TimeSeries &ts) {
  return integrator->match(ts, *this->collection);
}
